#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <Windows.h>

// Define this to turn on error checking
#define CUDA_ERROR_CHECK

#define CudaSafeCall( err ) __cudaSafeCall( err, __FILE__, __LINE__ )
#define CudaCheckError()    __cudaCheckError( __FILE__, __LINE__ )

inline void __cudaSafeCall( hipError_t err, const char *file, const int line )
{
#ifdef CUDA_ERROR_CHECK
	if ( hipSuccess != err )
	{
		fprintf( stderr, "cudaSafeCall() failed at %s:%i : %s\n",
			file, line, hipGetErrorString( err ) );
		exit( -1 );
	}
#endif

	return;
}

inline void __cudaCheckError(const char *file, const int line)
{
#ifdef CUDA_ERROR_CHECK
	hipError_t err = hipGetLastError();
	if (hipSuccess != err)
	{
		fprintf(stderr, "cudaCheckError() failed at %s:%i : %s\n",
			file, line, hipGetErrorString(err));
		exit(-1);
	}

	// More careful checking. However, this will affect performance.
	// Comment away if needed.
	err = hipDeviceSynchronize();
	if (hipSuccess != err)
	{
		fprintf(stderr, "cudaCheckError() with sync failed at %s:%i : %s\n",
			file, line, hipGetErrorString(err));
		exit(-1);
	}
#endif

	return;
}


hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void gol_step_kernel(bool *grid_start, bool* grid_tmp/*, size_t width, size_t height*/)
{
    size_t x = blockIdx.x*blockDim.x + threadIdx.x;
	size_t y = blockIdx.y*blockDim.y + threadIdx.y;

	size_t width = blockDim.x * gridDim.x;
	size_t height = blockDim.y * gridDim.y;

	//r�cup�rer les voisins sur grid start

	//Mettre � jour l�tat en x,y sur grid_tmp

    //c[i] = a[i] + b[i];
}

int main()
{
	size_t nb_loop = 100;
	size_t width, height;

	bool* _cpu_pointer,

	bool* grid_const;
	bool* grid_computed;
	CudaSafeCall(hipMalloc(&grid_const, width*height*sizeof(bool)));
	CudaSafeCall(hipMalloc(&grid_computed, width*height*sizeof(bool)));

	CudaSafeCall(hipMemcpy(grid_const, _cpu_pointer, width*height*sizeof(bool), hipMemcpyHostToDevice));

	dim3 grid_size = dim3(width/8, height/8);
	dim3 block_size = dim3(8, 8);

	for (int i = 0; i < nb_loop; ++i){
		gol_step_kernel <<< grid_size, block_size >>> (grid_const, grid_computed);
		auto tmp = grid_computed;
		grid_computed = grid_const;
		grid_const = tmp;
	}


	CudaSafeCall(hipMemcpy(_cpu_pointer, grid_const, width*height*sizeof(bool), hipMemcpyDeviceToHost));

	CudaSafeCall(hipFree(grid_const));
	CudaSafeCall(hipFree(grid_computed));

    return 0;
}
